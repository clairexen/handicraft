
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

void cudaCheckError_(hipError_t err, const char *filename, int linenr)
{
	if (err != hipSuccess) {
		fprintf(stderr, "Cuda error '%s' in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		abort();
	}
}

#define cudaCheckError(expr_) cudaCheckError_(expr_, __FILE__, __LINE__)

__global__
void my_memcpy_test_1(const int *src, int *dst)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	dst[idx] = src[idx];
}

__global__
void my_memcpy_test_2(const int *src, int *dst)
{
	int idx = 32 * (threadIdx.x + blockIdx.x * blockDim.x);
	for (int i = 0; i < 32; i++)
		dst[idx + i] = src[idx + i];
}

__global__
void my_memcpy_test_3(const int *src, int *dst)
{
	int idx = 32 * (threadIdx.x + blockIdx.x * blockDim.x);
	for (int i = 0; i < 32; i++)
		dst[idx + i] = __ldg(&src[idx + i]);
}

int main()
{
	int *buffer_a, *buffer_b;

	/* setup */

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	/* allocate */

	cudaCheckError(hipMalloc((void**)&buffer_a, sizeof(int)*1024*1024));
	cudaCheckError(hipMalloc((void**)&buffer_b, sizeof(int)*1024*1024));

	/* start collecting profiling data */

	cudaCheckError(hipProfilerStart());

	/* run test #1 */

	my_memcpy_test_1<<<1024, 1024>>>(buffer_a, buffer_b);
	cudaCheckError(hipDeviceSynchronize());

	/* run test #2 */

	my_memcpy_test_2<<<1024, 32>>>(buffer_a, buffer_b);
	cudaCheckError(hipDeviceSynchronize());

	/* run test #3 */

	my_memcpy_test_3<<<1024, 32>>>(buffer_a, buffer_b);
	cudaCheckError(hipDeviceSynchronize());

	/* done collecting profiling data */

	cudaCheckError(hipProfilerStop());

	/* free */

	cudaCheckError(hipFree(buffer_a));
	cudaCheckError(hipFree(buffer_b));

	return 0;
}

